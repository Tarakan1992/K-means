#include <mpi.h>
#include <hip/hip_runtime.h>
#include "clustering.h"
#include "clustering_io.h"

#define FILE_NAME_INDEX  1
#define ITEMS_COUNT_INDEX  2
#define PARAMS_COUNT_INDEX  3
#define CLUSTERS_COUNT_INDEX 4


int main(int argc, char** argv)
{
	int rank, size;
	int itemsCount, paramsCount, clustersCount;
	int itemsPerProc, offset;
	int ierr;
	char* filename;

    MPI_Init(&argc, &argv);

    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    //read cl parameters
    if(argc < 5)
    {
        if(rank == 0)
        {
            printf("Parameters must be {file} {number of items} {number of params} {number of clusters}\n");    
        }

    	MPI_Finalize();
        exit(1);
    }

   	itemsCount = atoi(argv[ITEMS_COUNT_INDEX]);
    paramsCount = atoi(argv[PARAMS_COUNT_INDEX]);
    clustersCount = atoi(argv[CLUSTERS_COUNT_INDEX]);
    filename = argv[FILE_NAME_INDEX];

    if(itemsCount < 1 || paramsCount < 1 || clustersCount < 1)
    {
        if(rank == 0)
        {
            printf("INVALID PARAMETERS");
        }
    	MPI_Finalize();
        exit(2);
    }

    //items per mpi-process
    itemsPerProc = (itemsCount + size - 1) / size;

    //calc file offset
    offset = itemsPerProc * rank;

    //last process correction
    if(rank == size - 1)
    {
        itemsPerProc = itemsPerProc - 1;
    }


    kMeans(filename, itemsPerProc, offset, clustersCount, paramsCount, size, rank);


    MPI_Finalize();
    return 0;
}

           